#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <random>
#include <chrono>
#include <iostream>
#include <thread>
#include <vector>

constexpr int N = 100000000;
constexpr int ITERATIONS = 20;
constexpr int BLOCK_SIZE = 1024;
constexpr int BLOCK_SIZE_X = 8;
constexpr int BLOCK_SIZE_Y = 8;
constexpr int BLOCK_SIZE_Z = 8;

#define CUDA_CHECK(call)                                                 \
    {                                                                    \
        hipError_t err = call;                                          \
        if (err != hipSuccess) {                                        \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ \
                      << " - " << hipGetErrorString(err) << std::endl;  \
            exit(EXIT_FAILURE);                                          \
        }                                                                \
    }

void CPU_add(float* a, float* b, float* c, size_t n) {
    for (size_t i = 0; i < n; i++) {
        c[i] = a[i] + b[i];
    }
}

// Multi-threaded CPU addition
void CPU_add_multithreaded(float* a, float* b, float* c, size_t n, int num_threads) {
    int block_size = n  / num_threads;
    std::vector<std::thread>threads;
    for (int i = 0;i < num_threads;i++)
    {
        int start = i * block_size;
        int end = (i == num_threads - 1) ? n  : start + block_size ;
        auto worker = [&](int start, int end) {
            for (int j = start;j < end;j++)
            {
                c[j] = a[j] + b[j];
            }
            };
        threads.emplace_back(worker, start, end);

    }
    for (auto& i : threads)
    {
        i.join();
    }
}

__global__ void GPU_add(float* a, float* b, float* c, size_t n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

__global__ void GPU_add_3D(float* a, float* b, float* c, size_t n) {
    int block_id = blockIdx.x + (gridDim.x * gridDim.y) * blockIdx.z + gridDim.x * (blockIdx.y);
    int block_offset = block_id * blockDim.x * blockDim.y * blockDim.z;
    int thread_offset = threadIdx.x + (blockDim.x * blockDim.y) * threadIdx.z + blockDim.x * (threadIdx.y);
    int i = block_offset + thread_offset;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    float* h_a = new float[N];
    float* h_b = new float[N];
    float* h_c = new float[N];
    float* h_c_gpu = new float[N];
    float* h_c_mt = new float[N];
    float* h_c_gpu_3d = new float[N];

    std::random_device rd;
    std::mt19937 rng(rd());
    std::uniform_real_distribution<float> dist(0.0f, 100.0f);

    for (int i = 0; i < N; ++i) {
        h_a[i] = dist(rng);
        h_b[i] = dist(rng);
    }

    float* d_a, * d_b, * d_c, *d_c_3d;
    size_t size = N * sizeof(float);
    CUDA_CHECK(hipMalloc(&d_a, size));
    CUDA_CHECK(hipMalloc(&d_b, size));
    CUDA_CHECK(hipMalloc(&d_c, size));
    CUDA_CHECK(hipMalloc(&d_c_3d, size));


    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    int GRID_SIZE = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    int nx = 1000, ny = 1000, nz = 100;
    dim3 BLOCK_SIZE_3D(BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z);
    dim3 GRID_SIZE_3D(
        (nx + BLOCK_SIZE_3D.x - 1) / BLOCK_SIZE_3D.x,
        (ny + BLOCK_SIZE_3D.y - 1) / BLOCK_SIZE_3D.y,
        (nz + BLOCK_SIZE_3D.z - 1) / BLOCK_SIZE_3D.z
    );


    printf("Performing warm-up runs...\n");
    for (int i = 0; i < 3; i++) {
        
        GPU_add << <GRID_SIZE, BLOCK_SIZE >> > (d_a, d_b, d_c, N);
        GPU_add_3D << <GRID_SIZE_3D, BLOCK_SIZE_3D >> > (d_a, d_b, d_c_3d, N);
        CUDA_CHECK(hipDeviceSynchronize());
    }

    // Single-threaded CPU benchmarking
    double cpu_seq_total_duration = 0.0;
    for (int iter = 0; iter < ITERATIONS; ++iter) {
        auto start = std::chrono::high_resolution_clock::now();
        CPU_add(h_a, h_b, h_c, N);
        auto end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> duration = end - start;
        cpu_seq_total_duration += duration.count();
    }
    double cpu_seq_avg_duration = cpu_seq_total_duration / ITERATIONS;
    std::cout << "Avg. time for single-threaded CPU_add: " << cpu_seq_avg_duration << " sec\n";

    // Multi-threaded CPU benchmarking
    int max_threads = std::thread::hardware_concurrency() / 2;
    double cpu_mt_total_duration = 0.0;
    for (int iter = 0; iter < ITERATIONS; ++iter) {
        auto start = std::chrono::high_resolution_clock::now();
        CPU_add_multithreaded(h_a, h_b, h_c_mt, N, max_threads);
        auto end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> duration = end - start;
        cpu_mt_total_duration += duration.count();
    }
    double cpu_mt_avg_duration = cpu_mt_total_duration / ITERATIONS;
    std::cout << "Avg. time for multi-threaded CPU_add (" << max_threads << " threads): "
        << cpu_mt_avg_duration << " sec\n";

    // GPU benchmarking
    double gpu_total_duration = 0.0;
    for (int iter = 0; iter < ITERATIONS; ++iter) {
        auto start = std::chrono::high_resolution_clock::now();
        GPU_add << <GRID_SIZE, BLOCK_SIZE >> > (d_a, d_b, d_c, N);
        CUDA_CHECK(hipDeviceSynchronize());
        auto end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> duration = end - start;
        gpu_total_duration += duration.count();
    }
    double gpu_avg_duration = gpu_total_duration / ITERATIONS;
    std::cout << "Avg. time for GPU_add: " << gpu_avg_duration << " sec\n";

    // GPU_3D benchmarking
    double gpu_3d_total_duration = 0.0;
    for (int iter = 0; iter < ITERATIONS; ++iter) {
        auto start = std::chrono::high_resolution_clock::now();
        GPU_add_3D << <GRID_SIZE_3D, BLOCK_SIZE_3D >> > (d_a, d_b, d_c_3d, N);
        CUDA_CHECK(hipDeviceSynchronize());
        auto end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> duration = end - start;
        gpu_3d_total_duration += duration.count();
    }
    double gpu_3d_avg_duration = gpu_3d_total_duration / ITERATIONS;
    std::cout << "Avg. time for GPU_add_3D: " << gpu_3d_avg_duration << " sec\n";




    // Speedup calculations
    std::cout << "Speedup (GPU vs single-threaded CPU): " << cpu_seq_avg_duration / gpu_avg_duration << '\n';
    std::cout << "Speedup (GPU vs multi-threaded CPU): " << cpu_mt_avg_duration / gpu_avg_duration << '\n';
    std::cout << "Speedup (Multi-threaded CPU vs single-threaded CPU): "<< cpu_seq_avg_duration / cpu_mt_avg_duration << '\n';
    std::cout << "Speedup (GPU vs GPU_3D): " << gpu_3d_avg_duration / gpu_avg_duration << '\n';

    // Verify results
    hipMemcpy(h_c_gpu, d_c, size, hipMemcpyDeviceToHost);
    hipMemcpy(h_c_gpu_3d, d_c_3d, size, hipMemcpyDeviceToHost);
    bool correct = true;
    for (int i = 0; i < N; i++) {
        if (fabs(h_c[i] - h_c_gpu[i]) > 1e-5 || fabs(h_c_mt[i] - h_c[i]) > 1e-5 || fabs(h_c[i] - h_c_gpu_3d[i]) > 1e-5) {
            correct = false;
            break;
        }
    }
    std::cout << "Results are " << (correct ? "correct" : "incorrect") << '\n';

    

    // Clean up
    delete[] h_a;
    delete[] h_b;
    delete[] h_c;
    delete[] h_c_mt;
    delete[] h_c_gpu;
    delete[] h_c_gpu_3d;
    CUDA_CHECK(hipFree(d_a));
    CUDA_CHECK(hipFree(d_b));
    CUDA_CHECK(hipFree(d_c));
    CUDA_CHECK(hipFree(d_c_3d));

    return 0;
}
