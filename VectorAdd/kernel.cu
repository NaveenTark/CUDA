﻿
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <random>
#include <chrono>
#include <iostream>

#define N 1000000
#define ITERATIONS 20


void CPU_add(float* a, float* b, float* c, size_t n)
{
	for (size_t i = 0;i < n;i++)
	{
		c[i] = a[i] + b[i];
	}
}



int main()
{
	float* h_a = new float[N];
	float* h_b = new float[N];
	float* h_c = new float[N];

	std::random_device rd; // Obtain a random number from hardware
	std::mt19937 rng(rd()); // Seed the generator
	std::uniform_real_distribution<float> dist(0.0f, 100.0f);

	// Initialize the array with random floats
	for (int i = 0; i < N; ++i) {
		h_a[i] = dist(rng); // Generate random float
		h_b[i] = dist(rng);
	}

	
	// Benchmarking CPU_add over multiple iterations
	double total_duration = 0.0;

	for (int iter = 0; iter < ITERATIONS; ++iter) {
		auto start = std::chrono::high_resolution_clock::now();
		CPU_add(h_a, h_b, h_c, N);
		auto end = std::chrono::high_resolution_clock::now();

		// Calculate the duration for this iteration
		std::chrono::duration<double> duration = end - start;
		total_duration += duration.count();
	}

	// Calculate average time taken
	double average_duration = total_duration / ITERATIONS;
	std::cout << "Average time taken for CPU_add over " << ITERATIONS
		<< " iterations: " << average_duration << " seconds" << std::endl;




	// Clean up
	delete[] h_a;
	delete[] h_b;
	delete[] h_c;

	return 0;
}