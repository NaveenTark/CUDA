#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <random>
#include <chrono>


constexpr int ITERATIONS = 20;
constexpr size_t M = 1024; // Number of rows in A and C
constexpr size_t K = 2048; // Number of columns in A and rows in B
constexpr size_t N = 1024; // Number of columns in B and C

constexpr int BLOCK_SIZE_X = 32;
constexpr int BLOCK_SIZE_Y = 32;
#define CUDA_CHECK(call)                                                 \
    {                                                                    \
        hipError_t err = call;                                          \
        if (err != hipSuccess) {                                        \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ \
                      << " - " << hipGetErrorString(err) << std::endl;  \
            exit(EXIT_FAILURE);                                          \
        }                                                                \
    }
void Basic_mul(float* a, float* b, float* out, int R, int C, int Z)
{
	for (int i = 0;i < R;i++)
	{
		for (int k = 0; k < Z;k++)
		{
			for (int j = 0;j < C;j++)
			{
				out[i * Z + k] += a[i * C + j] * b[j * Z + k];
			}
		}
	}
}
//Store sum in register and update output infrequently to avoid memory access
void Basic_mul_opt1(float* a, float* b, float* out, int R, int C, int Z)
{
	for (int i = 0;i < R;i++)
	{
		for (int k = 0; k < Z;k++)
		{
			float sum = 0.0f;
			for (int j = 0;j < C;j++)
			{
				sum += a[i * C + j] * b[j * Z + k];
			}
			out[i * Z + k] = sum;
		}
	}
}
//cache friendly access from B
void Basic_mul_opt2(float* a, float* b, float* out, int R, int C, int Z)
{
	for (int i = 0;i < R;i++)
	{
		for (int j = 0;j < C;j++)
		
		{
			float a_ij = a[i * C + j];
			for (int k = 0; k < Z;k++)
			{
				out[i * Z + k]+= a_ij * b[j * Z + k];
			}
			
		}
	}
}

__global__ void GPU_mul(float* a, float* b, float* c, int R, int C, int Z) {
	int row = blockDim.y * blockIdx.y + threadIdx.y;
	int col = blockDim.x * blockIdx.x + threadIdx.x;
	//This also works
	//int row = threadIdx.x + blockDim.y * threadIdx.y;
	//int col = blockIdx.x + gridDim.y * blockIdx.y;   
	
	if ((row < R) && (col < Z))
	{
		float sum = 0.0f;
		for (int i = 0;i < C;i++)
		{
			sum += a[row * C + i] * b[i * Z + col];
		}
		c[row * Z + col] = sum;
	}
	
}

int main()
{
	float* h_a = new float[M*K];
	float* h_b = new float[K*N];
	float* h_c_basic = new float[M*N]();
	float* h_c_basic_opt1 = new float[M * N]();
	float* h_c_basic_opt2 = new float[M * N]();
	float* h_c_gpu = new float[M * N]();


	std::random_device rd;
	std::mt19937 rng(rd());
	std::uniform_real_distribution<float> dist(0.0f, 100.0f);

	for (int i = 0; i < M * K; ++i) {
		h_a[i] = dist(rng);
	}
	for (int i = 0; i < K * N; ++i) {
		h_b[i] = dist(rng);
	}

	// Single-threaded CPU benchmarking
	double cpu_seq_total_duration = 0.0;
	for (int iter = 0; iter < ITERATIONS; ++iter) {
		auto start = std::chrono::high_resolution_clock::now();
		Basic_mul(h_a, h_b, h_c_basic, M,K,N);
		auto end = std::chrono::high_resolution_clock::now();
		std::chrono::duration<double> duration = end - start;
		cpu_seq_total_duration += duration.count();
	}
	double cpu_seq_avg_duration = cpu_seq_total_duration / ITERATIONS;
	std::cout << "Avg. time for single-threaded CPU basic mul: " << cpu_seq_avg_duration << " sec\n";

	// Single-threaded CPU benchmarking optimization 1
	double cpu_seq_total_duration1 = 0.0;
	for (int iter = 0; iter < ITERATIONS; ++iter) {
		auto start = std::chrono::high_resolution_clock::now();
		Basic_mul(h_a, h_b, h_c_basic_opt1, M, K, N);
		auto end = std::chrono::high_resolution_clock::now();
		std::chrono::duration<double> duration = end - start;
		cpu_seq_total_duration1 += duration.count();
	}
	double cpu_seq_avg_duration1 = cpu_seq_total_duration1 / ITERATIONS;
	std::cout << "Avg. time for single-threaded CPU basic mul optimization 1: " << cpu_seq_avg_duration1 << " sec\n";

	// Single-threaded CPU benchmarking optimization 2
	double cpu_seq_total_duration2 = 0.0;
	for (int iter = 0; iter < ITERATIONS; ++iter) {
		memset(h_c_basic_opt2, 0, M * N * sizeof(float));
		auto start = std::chrono::high_resolution_clock::now();
		Basic_mul(h_a, h_b, h_c_basic_opt2, M, K, N);
		auto end = std::chrono::high_resolution_clock::now();
		std::chrono::duration<double> duration = end - start;
		cpu_seq_total_duration2 += duration.count();
	}
	double cpu_seq_avg_duration2 = cpu_seq_total_duration2 / ITERATIONS;
	std::cout << "Avg. time for single-threaded CPU basic mul optimization 2: " << cpu_seq_avg_duration2 << " sec\n";

	
	dim3 BLOCK_SIZE_2D(BLOCK_SIZE_X, BLOCK_SIZE_Y);
	dim3 GRID_SIZE_2D(
		(N + BLOCK_SIZE_2D.x - 1) / BLOCK_SIZE_2D.x,
		(M + BLOCK_SIZE_2D.y - 1) / BLOCK_SIZE_2D.y
		
	);

	float* d_a, * d_b, * d_c;
	
	CUDA_CHECK(hipMalloc(&d_a, M*K*sizeof(float)));
	CUDA_CHECK(hipMalloc(&d_b, K*N*sizeof(float)));
	CUDA_CHECK(hipMalloc(&d_c, M*N*sizeof(float)));

	hipMemcpy(d_a, h_a, M * K * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, K * N * sizeof(float), hipMemcpyHostToDevice);

	printf("Performing warm-up runs...\n");
	for (int i = 0; i < 3; i++) {

		GPU_mul << <GRID_SIZE_2D, BLOCK_SIZE_2D >> > (d_a, d_b, d_c,M,K,N );
		CUDA_CHECK(hipDeviceSynchronize());
	}

	// GPU  benchmarking
	double gpu_total_duration = 0.0;
	for (int iter = 0; iter < ITERATIONS; ++iter) {
		auto start = std::chrono::high_resolution_clock::now();
		GPU_mul << <GRID_SIZE_2D, BLOCK_SIZE_2D >> > (d_a, d_b, d_c, M,K,N);
		CUDA_CHECK(hipDeviceSynchronize());
		auto end = std::chrono::high_resolution_clock::now();
		std::chrono::duration<double> duration = end - start;
		gpu_total_duration += duration.count();
	}
	double gpu_avg_duration = gpu_total_duration / ITERATIONS;
	std::cout << "Avg. time for GPU_mul: " << gpu_avg_duration << " sec\n";
	// Verify results
	hipMemcpy(h_c_gpu, d_c, M * N * sizeof(float), hipMemcpyDeviceToHost);
	
	bool correct = true;
	for (int i = 0; i < M*N; i++) {
		if (fabs(h_c_basic_opt2[i] - h_c_gpu[i]) > 1e-5) {
			correct = false;
			break;
		}
	}
	std::cout << "Results are " << (correct ? "correct" : "incorrect") << '\n';

	// Speedup calculations
	std::cout << "Speedup (GPU vs single-threaded CPU): " << cpu_seq_avg_duration2 / gpu_avg_duration << '\n';

	delete[] h_a;
	delete[] h_b;
	delete[] h_c_basic;
	delete[] h_c_basic_opt1;
	delete[] h_c_basic_opt2;
	CUDA_CHECK(hipFree(d_a));
	CUDA_CHECK(hipFree(d_b));
	CUDA_CHECK(hipFree(d_c));



}
